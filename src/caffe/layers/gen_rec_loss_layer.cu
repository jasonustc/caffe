#include "hip/hip_runtime.h"
#include <cstring>
#include <vector>
#include <cfloat>

#include "caffe/layer.hpp"
#include "caffe/layer_factory.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/loss_layers.hpp"

namespace caffe{
	template <typename Dtype>
	void GenRecLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const Dtype* mu_data = bottom[0]->gpu_data();
		const Dtype* sigma_data = bottom[1]->gpu_data();
		const Dtype* x_data = bottom[2]->gpu_data();
		//put intemediate result of mu into buffer data
		Dtype* mu_buffer_data = mu_sigma_buffer_.mutable_gpu_data();
		//put intemediate result of sigma into buffer diff
		Dtype* sigma_buffer_data = mu_sigma_buffer_.mutable_gpu_diff();
		const int count = bottom[0]->count();
		caffe_gpu_sub(count, x_data, mu_data, mu_buffer_data);
		caffe_gpu_sqr(count, mu_buffer_data, mu_buffer_data);
		caffe_gpu_sqr(count, sigma_data, sigma_buffer_data);
		//put the div result into multiplier diff
		caffe_gpu_div(count, mu_buffer_data, sigma_buffer_data, sum_multiplier_.mutable_gpu_diff());
		//sum
		Dtype loss1;
		caffe_gpu_dot(count, sum_multiplier_.gpu_data(), sum_multiplier_.mutable_gpu_diff(), &loss1);
		//since the invert sqare of sigma is not needed any more, we can just put log\sigma into this 
		//memory again
		caffe_gpu_log(count, sigma_data, sigma_buffer_data);
		Dtype loss2;
		caffe_gpu_dot(count, sum_multiplier_.gpu_data(), sigma_buffer_data, &loss2);
		Dtype loss3 = Dtype(0.5) * Dtype(num_feats_) * log(2 * Dtype(PI));
		Dtype loss = loss1 + loss2 + loss3;
		top[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
	}

	template <typename Dtype>
	__global__ void genrec_loss_mu_backward_kernel(const int n, const Dtype coeff,
		const Dtype* mu_data, const Dtype* sigma_data, const Dtype* x_data, Dtype* mu_diff){
		CUDA_KERNEL_LOOP(index, n){
			mu_diff[index] = coeff * Dtype(-2) * (x_data[index] - mu_data[index])/
				(sigma_data[index] * sigma_data[index]);
		}
	}

	template <typename Dtype>
	__global__ void genrec_loss_sigma_backward_kernel(const int n, const Dtype coeff,
		const Dtype* mu_data, const Dtype* sigma_data, const Dtype* x_data, Dtype* sigma_diff){
		CUDA_KERNEL_LOOP(index, n){
			sigma_diff[index] = coeff * Dtype(-2) * (x_data[index] - mu_data[index]) *
				(x_data[index] - mu_data[index]) / pow(sigma_data[index], Dtype(3)) +
				coeff / sigma_data[index];
		}
	}

	template <typename Dtype>
	void GenRecLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom){
		Dtype coeff = top[0]->cpu_diff()[0] / bottom[0]->num();
		const Dtype* mu_data = bottom[0]->gpu_data();
		const Dtype* sigma_data = bottom[1]->gpu_data();
		const Dtype* x_data = bottom[2]->gpu_data();
		Dtype* mu_diff = bottom[0]->mutable_gpu_diff();
		Dtype* sigma_diff = bottom[1]->mutable_gpu_diff();
		const int count = bottom[0]->count();
		if (propagate_down[0]){
			genrec_loss_mu_backward_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, coeff, mu_data, sigma_data, x_data, mu_diff);
			CUDA_POST_KERNEL_CHECK;
		}
		if (propagate_down[1]){
			genrec_loss_sigma_backward_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, coeff, mu_data, sigma_data, x_data, sigma_diff);
			CUDA_POST_KERNEL_CHECK;
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(GenRecLossLayer);
}