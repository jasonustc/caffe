#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/random_layers.hpp"

namespace caffe{
	template <typename Dtype>
	void SamplingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const Dtype* mu_data = bottom[0]->gpu_data();
		const Dtype* sigma_data = bottom[1]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		const int count = bottom[0]->count();
		Dtype* gaussian_data = gaussian_value_.mutable_gpu_data();
		caffe_gpu_rng_gaussian(count, Dtype(0.), Dtype(1.), gaussian_data);
		caffe_gpu_mul(count, sigma_data, gaussian_data, top_data);
		caffe_gpu_add(count, mu_data, top_data, top_data);
	}

	template <typename Dtype>
	void SamplingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& top){
		const int count = bottom[0]->count();
		Dtype* mu_diff = bottom[0]->mutable_gpu_diff();
		Dtype* sigma_diff = bottom[1]->mutable_gpu_diff();
		const Dtype* top_diff = top[0]->gpu_diff();
		const Dtype* gaussian_data = gaussian_value_.gpu_data();
		if (propagate_down[0]){
			caffe_copy(count, top_diff, mu_diff);
		}
		if (propagate_down[1]){
			caffe_gpu_mul(count, top_diff, gaussian_data, sigma_diff);
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(SamplingLayer);
}
