/********************************************************************************
** Copyright(c) 2015 USTC Reserved.
** auth: Xu Shen
** mail: shenxu@mail.ustc.edu.cn
** date: 2015/10/4
** desc: DecodingRecurrentLayer(GPU)
*********************************************************************************/
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/sequence_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void DRecurrentLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // Hacky fix for test time... reshare all the shared blobs.
  // TODO: somehow make this work non-hackily.
	if (this->phase_ == TEST) {
		unrolled_net_->ShareWeightData();
	}

	unrolled_net_->ForwardPrefilled();
}

INSTANTIATE_LAYER_GPU_FORWARD(DRecurrentLayer);

}  // namespace caffe
