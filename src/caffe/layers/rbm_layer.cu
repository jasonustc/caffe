#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/common_layers.hpp"

namespace caffe{

	template <typename Dtype>
	__global__ void SigmoidForward(const int n, const Dtype* in, Dtype* out){
		CUDA_KERNEL_LOOP(index, n){
			out[index] = 1. / (1. + exp(-in[index]));
		}
	}

	template <typename Dtype>
	void RBMLayer<Dtype>::Gibbs_vhvh_gpu(){
		const Dtype* weight_data = this->blobs_[0]->gpu_data();
		const Dtype* h_bias_data = this->blobs_[1]->gpu_data();
		const Dtype* v_bias_data = this->blobs_[2]->gpu_data();
		Dtype* pos_h_data = pos_h_.mutable_gpu_data();
		Dtype* neg_h_data = neg_h_.mutable_gpu_data();
		Dtype* positive_state_h_data = positive_state_h_.mutable_gpu_data();
		Dtype* negative_state_v_data = negative_state_v_.mutable_gpu_data();
		const Dtype* pos_v_data = pos_v_.gpu_data();
		Dtype* neg_v_data = neg_v_.mutable_gpu_data();
		const int count_h = pos_h_.count();
		const int count_v = neg_v_.count();
		//prop up
		//h: M x N  v: M x K w: N x K
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
			pos_v_data, weight_data, (Dtype)0, pos_h_data);
		if (bias_term_){
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
				bias_multiplier_.gpu_data(), h_bias_data, (Dtype)1., pos_h_data);
		}
		//sigmoid activation
		SigmoidForward<Dtype> << <CAFFE_GET_BLOCKS(count_h), CAFFE_CUDA_NUM_THREADS >> >(
			count_h, pos_h_data, pos_h_data);
		//sampling
		caffe_gpu_rng_bernoulli<Dtype>(count_h, pos_h_data, positive_state_h_data);
		//prop down
		//h: M x N  v: M x K w: N x K
		//TODO: need to convert the data type of state_h to Dtype
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
			positive_state_h_data, weight_data, (Dtype)0., neg_v_data);
		if (bias_term_){
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, 1, (Dtype)1.,
				bias_multiplier_.gpu_data(), v_bias_data, (Dtype)1., neg_v_data);
		}
		//sigmoid activation
		SigmoidForward<Dtype> << <CAFFE_GET_BLOCKS(count_v), CAFFE_CUDA_NUM_THREADS >> >(
			count_v, neg_v_data, neg_v_data);
		//sampling 
		caffe_gpu_rng_bernoulli<Dtype>(count_v, neg_v_data, negative_state_v_data);

		//prop up again
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
			negative_state_v_data, weight_data, (Dtype)0, neg_h_data);
		if (bias_term_){
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
				bias_multiplier_.gpu_data(), h_bias_data, (Dtype)1., neg_h_data);
		}

		//sigmoid activation
		SigmoidForward<Dtype> << <CAFFE_GET_BLOCKS(count_h), CAFFE_CUDA_NUM_THREADS >> >(
			count_h, neg_h_data, neg_h_data);
	}

	template <typename Dtype>
	void RBMLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		//top[0] shares data with pos_h_ data
		Gibbs_vhvh_gpu();
		//output reconstruction loss
		if (top.size() > 1){
			const int count = bottom[0]->count();
			const Dtype* bottom_data = bottom[0]->gpu_data();
			const Dtype* neg_v_data = neg_v_.gpu_data();
			Dtype* tmp_data = neg_v_.mutable_gpu_diff();
			caffe_gpu_sub<Dtype>(count, bottom_data, neg_v_data, tmp_data);
			Dtype loss;
			caffe_gpu_dot<Dtype>(count, tmp_data, tmp_data, &loss);
			top[1]->mutable_cpu_data()[0] = loss / bottom[0]->num();
		}
	}

	template <typename Dtype>
	void RBMLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom){
		//put positive data into buf data
		Dtype* pos_ass_data = weight_diff_buf_.mutable_gpu_data();
		//put negative data into buf diff
		Dtype* neg_ass_data = weight_diff_buf_.mutable_gpu_diff();
		const Dtype* pos_v_data = bottom[0]->gpu_data();
		const Dtype* pos_h_data = pos_h_.gpu_data();
		const Dtype* neg_v_data = neg_v_.gpu_data();
		const Dtype* neg_h_data = neg_h_.gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
		const Dtype* weight_data = this->blobs_[0]->gpu_data();
		Dtype scale = Dtype(1.) / bottom[0]->num();

		//Gradient with respect to weight
		if (this->param_propagate_down_[0]){
			caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
				pos_h_data, pos_v_data, (Dtype)0., pos_ass_data);
			caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
				neg_h_data, neg_v_data, (Dtype)0., neg_ass_data);
			caffe_gpu_sub(N_ * K_, pos_ass_data, neg_ass_data, neg_ass_data);
			//average by batch size
			caffe_gpu_axpby<Dtype>(this->blobs_[0]->count(), scale, neg_ass_data, 
				Dtype(1.), weight_diff);
		}

		//Gradient with respect to h_bias
		const int count_h = pos_h_.count();
		Dtype* h_bias_diff = this->blobs_[1]->mutable_gpu_diff();
		//\delta c_j = \delta c_j + p_h_j^(0) - p_h_j^(k)
		if (this->param_propagate_down_[1]){
			//put buffer data in neg_h_.diff()
			//pos_h_ is shared with top[0], be carefully to use it in other place
			caffe_gpu_sub<Dtype>(count_h, pos_h_data, neg_h_data, neg_h_.mutable_gpu_diff());
			//put intemediate result into neg_h_ data
			//average by batch size
			caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, scale, neg_h_.gpu_diff(), 
				bias_multiplier_.gpu_data(),(Dtype)1., h_bias_diff);
		}

		//Gradient with respect to v_bias
		const int count_v = pos_v_.count();
		Dtype* v_bias_diff = this->blobs_[2]->mutable_gpu_diff();
		//\delta b_j = \delta b_j + v_j^(0) - v_j^(k)
		if (this->param_propagate_down_[2]){
			//put buffer data in neg_v_.diff()
			//pos_v_ is shared with bottom[0], be carefully to use it in other place
			caffe_gpu_sub<Dtype>(count_v, pos_v_data, neg_v_data, neg_v_.mutable_gpu_diff());
			//put intemediate result into neg_v_ data
			//average by batch size
			caffe_gpu_gemv<Dtype>(CblasTrans, M_, K_, scale, neg_v_.gpu_diff(), 
				bias_multiplier_.gpu_data(), (Dtype)1., v_bias_diff);
		}

		if (propagate_down[0]){
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
				top_diff, weight_data, (Dtype)0., bottom_diff);
		}
		LOG(INFO) <<"sqr_diff: " << this->blobs_[0]->sumsq_diff();
		LOG(INFO) <<"abs_diff: " << this->blobs_[0]->asum_diff();
		LOG(INFO) << "abs_data: " << this->blobs_[0]->asum_data();
		LOG(INFO) << "sqr_data: " << this->blobs_[0]->sumsq_data();
		LOG(INFO) << "sqr_top: " << top[0]->sumsq_data();
		LOG(INFO) << "abs_top: " << top[0]->asum_data();
		LOG(INFO) << "abs_top_diff: " << top[0]->asum_diff();
		LOG(INFO) << "sqr_top_diff: " << top[0]->sumsq_diff();
	}

	INSTANTIATE_LAYER_GPU_FUNCS(RBMLayer);
}
