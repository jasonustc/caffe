#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {


template <typename Dtype>
__global__ void DropoutForward(const int n, const Dtype* in,
    const Dtype* mask, const Dtype threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    Dtype* mask =
        static_cast<Dtype*>(rand_vec_.mutable_gpu_data());
	  if (this->drop_type_ == DropoutParameter_DROPTYPE_UNIFORM){
		caffe_gpu_rng_uniform(count, a_, b_, mask);
		DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, bottom_data, mask, a_, scale_, top_data);
	  }
	  else if (this->drop_type_ == DropoutParameter_DROPTYPE_GAUSSIAN){
		caffe_gpu_rng_gaussian(count, mu_, sigma_, mask);
		DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, bottom_data, mask, mu_ - sigma_, scale_, top_data);
	  }
	  else{
		caffe_gpu_rng_uniform(count, (Dtype)0., (Dtype)1., mask);
		DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, bottom_data, mask, threshold_, scale_, top_data);
	  }
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void DropoutBackward(const int n, const Dtype* in_diff,
    const Dtype* mask, const Dtype threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const Dtype* mask =
          static_cast<const Dtype*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
	  if (this->drop_type_ == DropoutParameter_DROPTYPE_GAUSSIAN){
		  DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
			CAFFE_CUDA_NUM_THREADS>>>(
			  count, top_diff, mask, mu_ - sigma_, scale_, bottom_diff);
	  }
	  else if (this->drop_type_ == DropoutParameter_DROPTYPE_UNIFORM){
		  DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
			CAFFE_CUDA_NUM_THREADS>>>(
			  count, top_diff, mask, a_, scale_, bottom_diff);
	  }
	  else{
		  // NOLINT_NEXT_LINE(whitespace/operators)
		  DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
			CAFFE_CUDA_NUM_THREADS>>>(
			  count, top_diff, mask, threshold_, scale_, bottom_diff);
	  }
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutLayer);


}  // namespace caffe
