#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layer_factory.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/loss_layers.hpp"

namespace caffe{

	template <typename Dtype>
	void LatentLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const int count = bottom[0]->count();
		const Dtype* mu_data = bottom[0]->gpu_data();
		const Dtype* sigma_data = bottom[1]->gpu_data();
		Dtype* log_sqr_sigma_data = log_square_sigma_.mutable_gpu_data();
		Dtype mu_sqr_sum;
		caffe_gpu_dot(count, mu_data, mu_data, &mu_sqr_sum);
		Dtype sigma_sqr_sum;
		caffe_gpu_dot(count, sigma_data, sigma_data, &sigma_sqr_sum);
		caffe_gpu_sqr(count, sigma_data, log_sqr_sigma_data);
		caffe_gpu_log(count, log_sqr_sigma_data, log_sqr_sigma_data);
		Dtype log_sigma_sum;
		caffe_gpu_dot(count, log_sqr_sigma_data, sum_multiplier_.gpu_data(), &log_sigma_sum);
		Dtype loss = (mu_sqr_sum + sigma_sqr_sum - log_sigma_sum ) / bottom[0]->num() / Dtype(2);
		loss -= bottom[0]->count() / bottom[0]->num() / Dtype(2);
		top[0]->mutable_cpu_data()[0] = loss;
	}

	template <typename Dtype>
	__global__ void LatentLoss_Backward_kernel(const int n, const Dtype coeff, const Dtype* sigma_data,
		Dtype* sigma_diff_data){
		CUDA_KERNEL_LOOP(index, n){
			//TODO: check if data is not zero
			//first load to local device memory to save some time
			Dtype sig = max(sigma_data[index], Dtype(FLT_MIN));
			sigma_diff_data[index] = coeff *(sig - 1 / sig);
		}
	}

	template <typename Dtype>
	void LatentLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom){
		const int count = bottom[0]->count();
		const Dtype* mu_data = bottom[0]->gpu_data();
		Dtype* mu_diff = bottom[0]->mutable_gpu_diff();
		const Dtype* sigma_data = bottom[1]->gpu_data();
		Dtype* sigma_diff = bottom[1]->mutable_gpu_diff();
		const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num();
		if (propagate_down[0]){
			caffe_gpu_axpby(count, alpha, mu_data, Dtype(0.), mu_diff);
		}
		if (propagate_down[1]){
			LatentLoss_Backward_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
				(count, alpha, sigma_data, sigma_diff);
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(LatentLossLayer);
}
