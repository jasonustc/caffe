#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidActivate_gpu(const int n, const Dtype* in, Dtype* out){
	CUDA_KERNEL_LOOP(index, n){
		out[index] = 1. / (1. + exp(-in[index]));
	}
}

template <typename Dtype>
__global__ void ReluActivate_gpu(const int n, const Dtype* in, Dtype* out){
	CUDA_KERNEL_LOOP(index, n){
		out[index] = in[index]> 0 ? in[index] : 0;
	}
}

template <typename Dtype>
inline void activate_gpu(const int n, const Dtype* in, Dtype* out,
	AdaptiveDropoutParameter_ActType act_type){
	switch (act_type){
	case caffe::AdaptiveDropoutParameter_ActType_SIGMOID:
		SigmoidActivate_gpu<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, in, out);
		break;
	case caffe::AdaptiveDropoutParameter_ActType_RELU:
		ReluActivate_gpu<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, in, out);
		break;
	default:
		LOG(FATAL) << "Unkown activate function.";
	}
}

template <typename Dtype>
__global__ void ad_axpb(const int n, const Dtype* in, Dtype* out,
	const Dtype alpha, const Dtype beta){
	CUDA_KERNEL_LOOP(index, n){
		out[index] = alpha * in[index] + beta;
	}
}

template <typename Dtype>
void AdaptiveDropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight_data = this->blobs_[0]->gpu_data();
  Dtype* prob_data = this->prob_vec_.mutable_gpu_data();
  unsigned int *rand_vec_data = this->rand_vec_.mutable_gpu_data();
  const int count_weight = this->blobs_[0]->count();
  const int count_prob = this->prob_vec_.count();
  //compute prob_weight_data from weight_data
  //prob_weight_data = alpha_ * weight_data + beta_
//  if (alpha_ != 1 && beta_ != 0){
//	  caffe_mult_and_add_scalar_gpu<Dtype> << <CAFFE_GET_BLOCKS(count_weight), CAFFE_CUDA_NUM_THREADS >> >
//		  (count_weight, weight_data, prob_weight_data, alpha_, beta_);
//  }
//  else{
//	  caffe_copy(count_weight, weight_data, this->prob_weight_.mutable_gpu_data());
//  }
  //prob_data = alpha * op(bottom_data) * (weight_data) + beta * prob_data
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
      bottom_data, weight_data, (Dtype)0., prob_data);
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
        bias_multiplier_.gpu_data(),
        this->blobs_[1]->gpu_data(), (Dtype)1., this->prob_vec_.mutable_gpu_data());
  }
  //prob_act = f(alpha*(pi * bottom + bias) + beta)
  ad_axpb<Dtype> << <CAFFE_GET_BLOCKS(count_prob), CAFFE_CUDA_NUM_THREADS >> >
	  (count_prob, prob_data, prob_data, alpha_, beta_);
  //activate probability
  activate_gpu<Dtype>(count_prob, prob_data, prob_data, this->prob_act_type_);
  //compute hidden units
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
      bottom_data, weight_data, (Dtype)0., unact_hidden_.mutable_gpu_data());
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
        bias_multiplier_.gpu_data(),
        this->blobs_[1]->gpu_data(), (Dtype)1., unact_hidden_.mutable_gpu_data());
  }
  activate_gpu(top[0]->count(), unact_hidden_.gpu_data(), top_data, this->hidden_act_type_);
  CUDA_POST_KERNEL_CHECK;
  if (this->phase_ == TRAIN){
	  //p[i] is the probability of r[i]=1
	  caffe_gpu_rng_bernoulli<Dtype>(count_prob, prob_vec_.gpu_data(), rand_vec_.mutable_gpu_data());
	  caffe_gpu_mul_b<Dtype>(count_prob, top[0]->gpu_data(), rand_vec_.gpu_data(), 
		  top[0]->mutable_gpu_data());
  }
  else{
	caffe_gpu_mul<Dtype>(count_prob, top[0]->gpu_data(), prob_data, top[0]->mutable_gpu_data());
  }
}

template<typename Dtype>
__global__ void SigmoidBackward_gpu(const int n, const Dtype* in_diff,
	const Dtype* unact_data, Dtype* out_diff){
	CUDA_KERNEL_LOOP(index, n){
		const Dtype sigmoid_x = 1. / (1. + exp(-unact_data[index]));
		out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x);
	}
}

template <typename Dtype>
__global__ void ReLUBackward_gpu(const int n, const Dtype* in_diff,
	const Dtype* in_data, Dtype* out_diff){
	CUDA_KERNEL_LOOP(index, n){
		out_diff[index] = in_diff[index] * (in_data[index] > 0);
	}
}

template <typename Dtype>
inline void ActBackward_gpu(const int n, const Dtype* in_diff,
	const Dtype* in_data, Dtype* out_diff, AdaptiveDropoutParameter_ActType act_type){
	switch (act_type)
	{
	case caffe::AdaptiveDropoutParameter_ActType_RELU:
		ReLUBackward_gpu<Dtype ><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
			n, in_diff, in_data, out_diff);
		break;
	case caffe::AdaptiveDropoutParameter_ActType_SIGMOID:
		SigmoidBackward_gpu<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
			n, in_diff, in_data, out_diff);
		break;
	default:
		LOG(FATAL) << "unknown act function type.";
		break;
	}
} 

template <typename Dtype>
__global__ void DropoutBackward_gpu(const int n, const Dtype* in_diff,
	const unsigned int* mask, const float scale, Dtype* out_diff){
	CUDA_KERNEL_LOOP(index, n){
		out_diff[index] = in_diff[index] * scale * mask[index];
	}
}

template <typename Dtype>
void AdaptiveDropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
	const int count_top = top[0]->count();
	Dtype* top_diff = top[0]->mutable_gpu_diff();
	Dtype* unact_hidden_diff = this->unact_hidden_.mutable_gpu_diff();
	//backward through dropout
	const unsigned int* rand_vec_data = this->rand_vec_.gpu_data();
	//top_diff = top_diff * rand_vec_data
	//		caffe_gpu_mul_b<Dtype>(count_top, top_diff, rand_vec_data, top_diff);
	DropoutBackward_gpu<Dtype> << < CAFFE_GET_BLOCKS(count_top), CAFFE_CUDA_NUM_THREADS >> >(
		count_top, top_diff, rand_vec_data, 2., prob_vec_.mutable_gpu_diff());
	//backward through non-linear activation
	const Dtype* in_data = unact_hidden_.gpu_data();
	ActBackward_gpu(count_top, prob_vec_.gpu_diff(), in_data, unact_hidden_diff, hidden_act_type_);

	if (this->param_propagate_down_[0]) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		// Gradient with respect to weight
		caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
			unact_hidden_diff, bottom_data, (Dtype)0., this->blobs_[0]->mutable_gpu_diff());
	}
	if (bias_term_ && this->param_propagate_down_[1]) {
		// Gradient with respect to bias
		caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., unact_hidden_diff,
			bias_multiplier_.gpu_data(), (Dtype)0.,
			this->blobs_[1]->mutable_gpu_diff());
	}
	if (propagate_down[0]) {
		// Gradient with respect to bottom data
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
			unact_hidden_diff, this->blobs_[0]->gpu_data(), (Dtype)0.,
			bottom[0]->mutable_gpu_diff());
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(AdaptiveDropoutLayer);

}  // namespace caffe
