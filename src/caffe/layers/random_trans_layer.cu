#include "hip/hip_runtime.h"
/********************************************************************************
** Copyright(c) 2015 USTC Reserved.
** auth: Xu Shen
** mail: shenxu@mail.ustc.edu.cn
** date: 2015/11/4
** desc: RandomTransformLayer(GPU)
*********************************************************************************/
#include <vector>
#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/filler.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/random_layers.hpp"

namespace caffe{
	template <typename Dtype>
	void RandomTransformLayer<Dtype>::GetTransCoord_gpu(){
		//here we use cpu to compute tranform matrix
		float* tmat_cpu_data = tmat_.mutable_cpu_data();
		switch (sample_type_){
		case RandTransformParameter_SampleType_UNIFORM:
			if (rotation_){
				//randomly generate rotation angle
				caffe_rng_uniform(1, start_angle_, end_angle_, &curr_angle_);
				TMatFromParam(ROTATION, curr_angle_, curr_angle_, tmat_cpu_data);
			}
			if (scale_){
				caffe_rng_uniform(1, start_scale_, end_scale_, &curr_scale_);
				TMatFromParam(SCALE, curr_scale_, curr_scale_, tmat_cpu_data);
			}
			if (shift_){
				float shift_pixels_x = dx_prop_ * Width_;
				float shift_pixels_y = dy_prop_ * Height_;
				caffe_rng_uniform(1, -shift_pixels_x, shift_pixels_x, &curr_shift_x_);
				caffe_rng_uniform(1, -shift_pixels_y, shift_pixels_y, &curr_shift_y_);
				TMatFromParam(SHIFT, curr_shift_x_, curr_shift_y_, tmat_cpu_data);
			}
		//TODO: check if the threshold of the parameters are reasonable
		case RandTransformParameter_SampleType_GAUSSIAN:
			if (rotation_){
				//clip to in [-180, 180]
				caffe_rng_gaussian(1, (Dtype)0., std_angle_, &curr_angle_);
				curr_angle_ = curr_angle_ > -180 ? curr_angle_ : -180;
				curr_angle_ = curr_angle_ < 180 ? curr_angle_ : 180;
				TMatFromParam(ROTATION, curr_angle_, curr_angle_, tmat_cpu_data);
			}
			if (scale_){
				caffe_rng_gaussian(1, (Dtype)1., std_scale_, &curr_scale_);
				//clip to be in [min_scale_, max_scale_]
				curr_scale_ = curr_scale_ > min_scale_ ? curr_scale_ : min_scale_;
				curr_scale_ = curr_scale_ < max_scale_ ? curr_scale_ : max_scale_;
				TMatFromParam(SCALE, curr_scale_, curr_scale_, tmat_cpu_data);
			}
			if (shift_){
				Dtype shift_std_x = std_dx_prop_ * Width_;
				Dtype shift_std_y = std_dy_prop_ * Height_;
				caffe_rng_gaussian(1, (Dtype)0., shift_std_x, &curr_shift_x_);
				caffe_rng_gaussian(1, (Dtype)0., shift_std_y, &curr_shift_y_);
				Dtype max_shift_pixels_width = max_shift_prop_ * Width_;
				Dtype max_shift_pixels_height = max_shift_prop_ * Height_;
				//clip shift proportion to be less or equal max_shift_prop_
				curr_shift_x_ = curr_shift_x_ < max_shift_pixels_width ? curr_shift_x_ : max_shift_pixels_width;
				curr_shift_x_ = curr_shift_x_ > (-max_shift_pixels_width) ? curr_shift_x_ : (-max_shift_pixels_width);
				curr_shift_y_ = curr_shift_y_ < max_shift_pixels_height ? curr_shift_y_ : max_shift_pixels_height;
				curr_shift_y_ = curr_shift_y_ > (-max_shift_pixels_height) ? curr_shift_y_ : (-max_shift_pixels_height);
				TMatFromParam(SHIFT, curr_shift_x_, curr_shift_y_, tmat_cpu_data);
			}
			break;
		default:
			LOG(FATAL) << "Unkown sampling type";
			break;
		}
		//Canoincal size is set, so after finding the transformation,
		//crop or pad to that canonical size.
		//First find the coordinate matrix for this transformation
		//here we don't change the shape of the input 2D map
		GenCoordMatCrop_gpu(tmat_, Height_, Width_, original_coord_, coord_idx_, BORDER_, INTERP_);
	}

	template <typename Dtype>
	void RandomTransformLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*> &top){
		const int count = bottom[0]->count();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		//if there are no random transformations, we just copy bottom data to top blob
		//in test phase, we don't do any transformations
		if ((!scale_ && !rotation_ && !shift_) || this->phase_ == TEST){
			caffe_copy(count, bottom_data, top_data);
		}
		else{
			//get coordinate map matrix
			GetTransCoord_gpu();
			//Apply Imterpolation on bottom_data using tmat_[i] into top_data.
			InterpImageNN_gpu(bottom[0], coord_idx_.gpu_data(), top[0], INTERP_);
		}
	}

	template <typename Dtype>
	void RandomTransformLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*> &bottom){
		const int count = top[0]->count();
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		//Reset bottom diff.
		caffe_gpu_set(count, Dtype(0.), bottom_diff);
		if (propagate_down[0]){
			if (!scale_ && !shift_ && !rotation_){
				caffe_copy(count, top_diff, bottom_diff);
			}
			else{
				BackPropagateErrorNN_gpu(top[0], coord_idx_.gpu_data(), bottom[0], INTERP_);
			}
		}
	}

	//since the atomicAdd gpu function in transform only support float,
	//so we only register float functions here
	INSTANTIATE_LAYER_GPU_FUNCS_FLOAT_ONLY(RandomTransformLayer);
}
