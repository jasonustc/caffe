#include "hip/hip_runtime.h"
//Implemented by Angjoo Kanzawa, Abhishek Sharma 2013
#include <cstdlib>
#include <cstring>
#include <stdio.h>
#include <thrust\device_vector.h>
#include <thrust\count.h>

#include "caffe/common.hpp"
#include "caffe/util/transform.hpp"
#include "caffe/blob.hpp"

namespace caffe{
	template <typename Dtype>
	__device__ void Reflect_gpu(Dtype& val, const int size){
		if (val < Dtype(0)){
			val = -floor(val);
			val = static_cast<Dtype>(static_cast<int>(val) % (2 * size - 1));
		}
		if (val >= size){
			val = 2 * size - 2 - val;
		}
	}

	template <typename Dtype>
	__device__ void Clamp_gpu(Dtype& val, const int size){
		val = max(static_cast<Dtype>(0.), min(val, static_cast<Dtype>(size - 1)));
	}

	template <typename Dtype>
	__global__ void generate_nn_coord_kernel(const int N, const int height, const int width,
		const int height_new, const int width_new, const Border& border,
		const float* coord_data_res, float* &coord_data){
		float old_cy = static_cast<float>(height - 1) / 2.;
		float old_cx = static_cast<float>(width - 1) / 2.;
		CUDA_KERNEL_LOOP(index, N){
			//nearest pixel
			float row = round(coord_data_res[3 * index] + old_cy);
			float col = round(coord_data_res[3 * index + 1] + old_cx);
			switch (border)
			{
			case CROP:
				if ((row >= height || row < 0) || (col >= width) || (col < 0)){
					coord_data[index] = Dtype(-1);
					continue;
				}
				break;
			case CLAMP:
				Clamp_gpu(row, height);
				Clamp_gpu(col, width);
				break;
			case REFLECT:
				Reflect_gpu(row, height);
				Reflect_gpu(col, width);
				break;
			default:
				break;
			}
			coord_data[index] = round(row) * width + round(col);
		}
	}

	template <typename Dtype>
	__global__ void generate_bilinear_coord_kernel(const int N, const int height, const int width,
		const int height_new, const int width_new, const Border border,
		const float* coord_data_res, float* coord_data){
		CUDA_KERNEL_LOOP(index, N){
			float old_cy = static_cast<float>(height - 1) / 2.;
			float old_cx = static_cast<float>(width - 1) / 2.;
			float row = coord_data_res[3 * index] + old_cy;
			float col = coord_data_res[3 * index + 1] + old_cx;
			//p00 => (r0, c0) p11 => (r1,c1)
			switch (border)
			{
			case CROP:
				//skip interpolation
				if ((row >= height - 0.5 || row < -0.5) || (col >= width - 0.5) || 
					(col < -0.5)){
					coord_data[index] = Dtype(-1);
					continue;
				}
				break;
			case CLAMP:
				Clamp_gpu(row, height);
				Clamp_gpu(col, width);
				break;
			case REFLECT:
				Reflect_gpu(row, height);
				Reflect_gpu(col, width);
				break;
			default:
				break;
			}
			//p00, trunc(x), trunc(y)
			float row0 = trunc(row);
			float col0 = trunc(col);
			//p11
			float row1 = trunc(row + 1) > (height - 1) ? height - 1 : trunc(row + 1);
			float col1 = trunc(col + 1) > (width - 1) ? width - 1 : trunc(col + 1);

			//if p00 is outside, don't compute difference
			float dc = col0 == col1 ? 0 : col - col0;
			float dr = row0 == row1 ? 0 : row - row0;

			//left up point
			coord_data[index] = row0 * width + col0;
			//right down point
			coord_data[index + N] = row1 * width + col1;
			//column difference
			coord_data[index + 2 * N] = dc;
			//row difference
			coord_data[index + 3 * N] = dr;
		}
	}

	//currently, we only support float here
	void GenCoordMatCrop_gpu(Blob<float>& tmat, const int height, const int width,
		Blob<float>& ori_coord, Blob<float>& coord_idx, const Border& border, const Interp& interp){
		float* tmat_cpu_data = tmat.mutable_cpu_data();
		CHECK(border == CLAMP || border == CROP || border == REFLECT) << 
			"Unknown border type: " << border;
		//transform to inverse new_image => ori_image
		Invert3x3(tmat_cpu_data);

		float cy = static_cast<float>(height - 1) / 2.;
		float cx = static_cast<float>(width - 1) / 2.;

		//substract center
		AddShift(-cy, -cx, tmat_cpu_data, LEFT);

		//we can use ori_coord data and diff for buffer of coordinates
		//since it is only used in this step
		const float *coord_data_tmp = ori_coord.gpu_data();
		float *coord_data_res = ori_coord.mutable_gpu_diff();
		float *tmat_gpu_data = tmat.mutable_gpu_data();

		//Apply transformation
		caffe_gpu_gemm<float>(CblasNoTrans, CblasNoTrans, height * width, 3, 3, 1.f,
			coord_data_tmp, tmat_gpu_data, 0.f, coord_data_res);

		//save the final result into coord_idx
		float *coord_data_final = coord_idx.mutable_gpu_data();
		int n = height * width;
		switch (interp)
		{
		case NN:
			generate_nn_coord_kernel<float><< <CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS >>>(
				n, height, width, height, width, border, coord_data_res, coord_data_final);
			break;
		case BILINEAR:
			generate_bilinear_coord_kernel<float><< <CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS >>>(
				n, height, width, height, width, border, coord_data_res, coord_data_final);
			break;
		default:
			LOG(FATAL) << "Unkown interpolation type " << interp;
			break;
		}
		CUDA_POST_KERNEL_CHECK;
	}

	template <typename Dtype>
	__global__ void nn_interpolation_kernel(const int nthreads, const Dtype *oldDPtr,
		const int oldSheetCount, Dtype* newDPtr,
		const int newSheetCount, const float* coord){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index < nthreads){
			int offset = index % newSheetCount;
			int numSheet = index / newSheetCount;
			int backSheetOffset = static_cast<int>(coord[offset]);
			if (backSheetOffset >= 0){
				newDPtr[numSheet * newSheetCount + offset] =
					oldDPtr[numSheet * oldSheetCount + backSheetOffset];
			}
			else{
				newDPtr[numSheet * newSheetCount + offset] = 0;
			}
		}
	}

	template <typename Dtype>
	__global__ void bilinear_interpolation_kernel(const int nthreads, const Dtype* oldDPtr,
		const int oldSheetCount, Dtype* newDPtr, const int newSheetCount, const float* coord,
		const int W){
		//need W
		//what is W here? maybe the width of transformed matrix?
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index < nthreads){
			int offset = index % newSheetCount; //p00: r0 * W + c0
			int numSheet = index / newSheetCount;
			int backSheetOffset = static_cast<int>(coord[offset]);
			if (backSheetOffset >= 0){
				int c0 = backSheetOffset % W;
				//p11: r1 * W + c1
				int ind_p11 = static_cast<int>(coord[offset + newSheetCount]);
				int c1 = ind_p11 % W;

				int ind_p01 = backSheetOffset - c0 + c1;//r0 * W + c1
				int ind_p10 = ind_p11 - c1 + c0; //r1 * W + c0

				float dc = coord[offset + 2 * newSheetCount];
				float dr = coord[offset + 3 * newSheetCount];

				float w00 = (1 - dc) * (1 - dr);
				float w01 = (1 - dr) * dc;
				float w10 = (1 - dc) * dr;
				float w11 = dr * dc;

				int bigOffset = numSheet * oldSheetCount;
				newDPtr[numSheet * newSheetCount + offset] =
					w00 * oldDPtr[bigOffset + backSheetOffset] +
					w01 * oldDPtr[bigOffset + ind_p01] +
					w10 * oldDPtr[bigOffset + ind_p10] +
					w11 * oldDPtr[bigOffset + ind_p11];
			}
			else{
				newDPtr[numSheet * newSheetCount + offset] = 0;
			}
		}
	}

	template <typename Dtype>
	void InterpImageNN_gpu(const Blob<Dtype>* orig, const float* coord,
		Blob<Dtype>* warped, const Interp& interp){
		//Get the paramters from the original and warped and apply the
		//transformation to it.
		const Dtype* orgDataPtr = orig->gpu_data();
		Dtype* warpedDataPtr = warped->mutable_gpu_data();
		int oldNPerSheet = orig->height() * orig->width();
		int newNPerSheet = warped->height() * warped->width();
		int nCount = warped->count();
		switch (interp)
		{
		case NN:
			nn_interpolation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
				CAFFE_CUDA_NUM_THREADS >> >(nCount, orgDataPtr, oldNPerSheet, 
				warpedDataPtr, newNPerSheet, coord);
			break;
		case BILINEAR:
			bilinear_interpolation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
				CAFFE_CUDA_NUM_THREADS >> >(nCount, orgDataPtr, oldNPerSheet, warpedDataPtr, 
				newNPerSheet, coord, orig->width());
			break;
		default:
			LOG(ERROR) << "Unkown interpolation mode " << interp;
			break;
		}
		CUDA_POST_KERNEL_CHECK;
	}

	//explicit instantiation
	template void InterpImageNN_gpu(const Blob<float>* orig, const float* coord,
		Blob<float>* warped, const Interp& interp);
//	template void InterpImageNN_gpu(const Blob<double>* orig, const float* coord,
//		Blob<double>* warped, const Interp& interp);

	/*
	 *******PropagateErrorNN_gpu********
	 *If we kernalize each pixel in the top(warped image), bc of race conditions
	 *we need to use atomaticAdd, but it's slow and there is no double implementation
	 *of atomicAdd.
	 *So instead, parallelize over each pixel in the bottom (original) and for each pixel
	 * loop over the coord, find those top neurons that came from this bottom pixel and add.
	 * Similar to MaxPoolBackward Super. fucking slow. duh.
	 */
	template <typename Dtype>
	__global__ void PropagateErrorNN_kernel_nonatomic(
		const int nthreads, const Dtype* top_diff, const int width,
		const int height, const int channels, const int num,
		const int top_len, const float* coord, Dtype* bottom_diff){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index < nthreads){
			//find out the target index to look for in coord
			//can do this the way abhishek did so we can save on some computation(like
			//with SheetCount)
			int w = index % width;
			int h = (index / width) % height;
			int c = (index / width / height) % channels;
			int n = index / width / height / channels;

			int target_ind = h * width + w;
			//move over top_diff ptr to the beginning of its hxw sheet:
			//top_len = width_top * height_top
			top_diff += (n * channels + c) * top_len;

			Dtype gradient = 0;
			//loop over coord and add to grad if coord[i] == target_ind
			for (int i = 0; i < top_len; ++i){
				gradient += top_diff[i] * (static_cast<int>(coord[i]) == target_ind);
			}
			bottom_diff[index] += gradient;
		}
	}

	template <typename Dtype>
	__global__ void nn_backpropagation_kernel(int nThreads, const Dtype* topDataPtr,
		int topSheetCount, Dtype* bottomDataPtr,
		int bottomSheetCount, const float* coord){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index < nThreads){
			int offset = index % topSheetCount;
			int numSheet = index / topSheetCount;

			int bottomSheetOffset = static_cast<int>(coord[offset]);
			if (bottomSheetOffset >= 0){
				int bottomFinalOffset = numSheet * bottomSheetCount + bottomSheetOffset;
				//AJ: as atomicAdd is only available to float, this only works if
				//Dtype = float
				atomicAdd((&bottomDataPtr[bottomFinalOffset]),
					static_cast<float>(topDataPtr[numSheet * topSheetCount + offset]));
			}
		}
	}

	template <typename Dtype>
	__global__ void bilinear_backpropagation_kernel(int nThreads, const Dtype* topDataPtr,
		int topSheetCount, Dtype* bottomDataPtr, int bottomSheetCount,
		const float* coord, int W){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index < nThreads){
			int offset = index % topSheetCount;
			int numSheet = index / topSheetCount;
			int bottomSheetOffset = static_cast<int>(coord[offset]);
			if (bottomSheetOffset >= 0){
				int c0 = bottomSheetOffset % W;
				int ind_p11 = static_cast<int>(coord[offset + topSheetCount]);
				int c1 = ind_p11 % W;

				int ind_p01 = bottomSheetOffset - c0 + c1; //r0 * W + c1
				int ind_p10 = ind_p11 - c1 + c0;

				float dc = coord[offset + 2 * topSheetCount];
				float dr = coord[offset + 3 * topSheetCount];

				float w00 = (1 - dc)*(1 - dr);
				float w01 = (1 - dr)*dc;
				float w10 = (1 - dc)*dr;
				float w11 = dr * dc;

				float top_error = static_cast<float>(topDataPtr[index]);

				int commonOffset = numSheet * bottomSheetCount;

				//p00
				atomicAdd((&bottomDataPtr[commonOffset + bottomSheetOffset]),
					w00 * top_error);
				//p01
				atomicAdd((&bottomDataPtr[commonOffset + ind_p01]), w01 * top_error);
				//p10
				atomicAdd((&bottomDataPtr[commonOffset + ind_p10]), w10 * top_error);
				//p11
				atomicAdd(&bottomDataPtr[commonOffset + ind_p11], w11 * top_error);
			}
		}
	}

	template <typename Dtype>
	void BackPropagateErrorNN_gpu(const Blob<Dtype>* top, const float* coord,
		Blob<Dtype>* bottom, const Interp &interp){
	    //Get the parameters from the original and warped and apply the 
		//transformation to it.
		const Dtype* topDataPtr = top->gpu_diff();
		Dtype* bottomDataPtr = bottom->mutable_gpu_diff();
		int topNPerSheet = top->height() * top->width();
		int bottomNPerSheet = bottom->height() * bottom->width();
		//atomicAdd needs nTop many threads
		int nCount = top->count();
		switch (interp)
		{
		case NN:
			nn_backpropagation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
				CAFFE_CUDA_NUM_THREADS >> >(nCount, topDataPtr,
				topNPerSheet, bottomDataPtr, bottomNPerSheet, coord);
			break;
		case BILINEAR:
			bilinear_backpropagation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
				CAFFE_CUDA_NUM_THREADS >> >(nCount, topDataPtr,
				topNPerSheet, bottomDataPtr, bottomNPerSheet, coord, bottom->width());
			break;
		default:
			LOG(ERROR) << "Unknown interpolation mode " << interp;
			break;
		}
		CUDA_POST_KERNEL_CHECK;
	}

	//explicit instantiation
	template void BackPropagateErrorNN_gpu(const Blob<float>* top, const float* coord,
		Blob<float>* bottom, const Interp &interp);
//	template void PropagateErrorNN_gpu(const Blob<double>* top, const float* coord,
//		Blob<double>* bottom, const Interp &interp);

}